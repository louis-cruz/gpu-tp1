#include "hip/hip_runtime.h"

#include <omp.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
#include <iostream>
#include <fstream>
#include <string>

using namespace std;


__global__ void deviceCudaReduction(float *blockSums, float *reducedSum) {

    __shared__ float sdata[256];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = blockSums[i];

    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {

        int index = 2 * s * threadIdx.x;;

        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + s];
        }

        __syncthreads();
    }

    if (tid == 0) {
      //reducedSum[blockIdx.x] = sdata[0];
      atomicAdd(reducedSum, sdata[0]);
      }
}

__global__ void deviceCudaBlockSum(float *blockSums, double *step, int *stepsPerThread){


        __shared__ float blockSum;
        if(threadIdx.x == 0)
          blockSum = 0;
      
        __syncthreads();

        int index = blockIdx.x * blockDim.x + threadIdx.x;

        float subSum = 0;
        for(int i = index * *stepsPerThread; i < (index + 1) * *stepsPerThread;i++){
          double x = (i-0.5)* *step;
          float sum = 4.0/(1.0+x*x);
          subSum += sum;
        }
        atomicAdd(&blockSum, subSum);

        __syncthreads();

        if(threadIdx.x == 0)
          blockSums[blockIdx.x] = blockSum;

        __syncthreads();
}

double calculatePiReduction(int num_steps, double step, int threads, int stepsPerThread){

    int blocks = num_steps / threads;
    blocks /= stepsPerThread;
/*
    int chunkSize = 1024;
    int chunkCount = blocks / chunkSize;*/

    // variables used for both stages
    float *dev_blockSums;
    float host_blockSums[blocks] = {0};
    int * dev_stepsPerThread;
    hipMalloc((void **) &dev_stepsPerThread, sizeof(int));
    hipMemcpy(dev_stepsPerThread, &stepsPerThread, sizeof(int), hipMemcpyHostToDevice);

    //------------------ stage1 ---------------------------------------//
    // define host and device variables
    double *dev_step;
    // gpu memory allocation and copy
    hipMalloc((void **) &dev_step, sizeof(double));
    hipMalloc((void **) &dev_blockSums, sizeof(float)*blocks);
	  hipMemcpy(dev_blockSums, host_blockSums, sizeof(float)*blocks, hipMemcpyHostToDevice);
	  hipMemcpy(dev_step, &step, sizeof(double), hipMemcpyHostToDevice);
    // run kernel to compute individual block sums
    deviceCudaBlockSum<<<blocks, threads>>>(dev_blockSums, dev_step, dev_stepsPerThread);
    // transfer data from gpu to main memory
	  hipMemcpy(host_blockSums,dev_blockSums, sizeof(float)*blocks, hipMemcpyDeviceToHost);
    // liberate memory
    hipFree(dev_blockSums);

    //----------------- stage 2 ----------------------------------------//
    // define host and device variables
    //float host_reduced[blocks] = {0};
    float *dev_reduced;
    float temp = 0;
    float *host_reduced =  &temp;
    // gpu memory allocation and copy
    hipMalloc((void **) &dev_reduced, sizeof(float));
    hipMalloc((void **) &dev_blockSums, sizeof(float)*blocks);
	  hipMemcpy(dev_blockSums, host_blockSums, sizeof(float)*blocks, hipMemcpyHostToDevice);
	  hipMemcpy(dev_reduced, host_reduced, sizeof(float), hipMemcpyHostToDevice);
    // run kernel to compute total block sum
    blocks = blocks/threads;
    deviceCudaReduction<<<blocks, threads, threads*sizeof(float)>>>(dev_blockSums, dev_reduced);
    // transfer data from gpu to main memeory
	  hipMemcpy(host_reduced, dev_reduced, sizeof(float), hipMemcpyDeviceToHost);
    // liberate memory
    hipFree(dev_blockSums);
    hipFree(dev_reduced);

    cout<< *host_reduced * step<< endl;
    //--------------------------------------------------------------//

    return step * *host_reduced;
}

